//nvcc -o hello_cudnn hello_cudnn.cu


#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h> 

#include <cudnn.h>


__global__ void print_from_gpu(void) {
	printf("Hello World! from thread [%d,%d] \
		From device\n", threadIdx.x,blockIdx.x); 
}

int main(void) { 
	printf("Hello World from host!\n"); 
	print_from_gpu<<<3,3>>>();
	hipDeviceSynchronize();
return 0; 
}
